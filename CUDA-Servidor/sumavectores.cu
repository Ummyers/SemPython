#include "hip/hip_runtime.h"

#include <stdio.h>

// Kernel (funcion) que se invoca desde el Host y se ejecuta en un dispositivo
__global__ void suma_vectores(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// Funcion auxiliar que encapsula la suma con CUDA
void suma_CUDA(int* c, const int* a, const int* b, int tam) {
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // Reservamos espacio de memoria para los datos, 2 de entrada y una salida
    hipMalloc((void**)&dev_c, tam * sizeof(int));
    hipMalloc((void**)&dev_a, tam * sizeof(int));
    hipMalloc((void**)&dev_b, tam * sizeof(int));

    // Copiamos los datos de entrada desde el CPU a la memoria del GPU
    hipMemcpy(dev_a, a, tam * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, tam * sizeof(int), hipMemcpyHostToDevice);

    // Se invoca al kernel en el GPU con un hilo por cada elemento
    // 2 es el numero de bloques y (tam + 1)/2 es el numero de hilos en cada bloque
    suma_vectores<<<2, (tam + 1) / 2>>>(dev_c, dev_a, dev_b, tam);
    
    // Esta funcion espera a que termine de ejecutarse el kernel y 
    // devuelve los errores que se hayan generado al ser invocado
    hipDeviceSynchronize();

    // Copiamos el vector resultado de la memoria del GPU al CPU
    hipMemcpy(c, dev_c, tam * sizeof(int), hipMemcpyDeviceToHost);

    // Se libera la memoria empleada
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

// Funcion principal que sirve de prueba para el algoritmo
int main(int argc, char** argv) {
    
    // Datos de entrada para nuestra funcion
    const int tam = 5;
    const int a[tam] = {  1,  2,  3,  4,  5 };
    const int b[tam] = { 10, 20, 30, 40, 50 };
    int c[tam] = { 0 };

    // Se llama a la funcion que encapsula el Kernel
    suma_CUDA(c, a, b, tam);

    // Mostramos resultado
    printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", c[0], c[1], c[2], c[3], c[4]);

    // Se liberan recursos
    hipDeviceReset();

    return 0;
}
