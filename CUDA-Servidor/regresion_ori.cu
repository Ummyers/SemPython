
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>


__host__ float barra(float* x, int n)
{
	int i;
	float xb = 0.0;
	for (i = 0;i < n;i++) {
		xb += x[i];
	}
	xb = xb / n;

	return xb;
}

__global__ void Saxx_device(float* x, float* c, float xb, int n)
{
	int i = threadIdx.x;
	if (i < n)
		c[i] = (x[i] - xb) * (x[i] - xb);

}

__global__ void Saxy_device(float* x, float* y, float* d, float xb, float yb, int n)
{
	int i = threadIdx.x;

	if (i < n)
		d[i] = (x[i] - xb) * (y[i] - yb);

}

int main()
{

	// cuerpo de las variables del host
	float* x;
	float* y;
	float h_xb;
	float h_yb;
	float* h_saxy;
	float* h_saxx;
	float sxx;
	float sxy;
	int size;
	// cuerpo de las variables del device
	float* c;
	float* d_saxx;
	float* d_saxy;
	float* d;
	float* d_x;
	float* d_y;


	// numero de elementos

	printf("ingrese\n");
	scanf("%d", &size);

	// alocamos las variables en el host

	x = (float*)malloc(sizeof(float) * size);
	y= (float*)malloc(sizeof(float) * size);

	// llenamos los vectores
	for (int i = 0;i < size;i++) {

		x[i] = rand() + (float)i * 0.01;
		y[i] = rand() - (float)i * 0.01;
	}

	// llamamos a barra para calcular el promedio

	h_xb= barra(x, size);
	h_yb= barra(y, size);

	// alocamos memoria en el device 

	hipMalloc((void**)&c, sizeof(float) * size);
	hipMalloc((void**)&d, sizeof(float) * size);
	hipMalloc((void**)&d_x, sizeof(float) * size);
	hipMalloc((void**)&d_y, sizeof(float) * size);

	// pasando el contenido al device

	hipMemcpy(d_x, x, sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(float) * size, hipMemcpyHostToDevice);



	// invocamos los kernel

	Saxx_device <<< 1, size >>> (d_x,c, h_xb, size);
	Saxy_device <<< 1, size >>> (d_x,d_y,d, h_xb, h_yb, size);


		// alocamos en el host

	h_saxx = (float*)malloc(sizeof(float) * size);
	h_saxy = (float*)malloc(sizeof(float) * size);

	// obtenemos del device los elementos

	hipMemcpy(h_saxx, c, sizeof(float) * size, hipMemcpyDeviceToHost);

	hipMemcpy(h_saxy, d, sizeof(float) * size, hipMemcpyDeviceToHost);


	//  llamamos a barra
	sxx = barra(h_saxx, size);
	sxy = barra(h_saxy, size);

	// declaramos variables de regresion
	float beta1G, beta0G;

	beta1G = sxy / sxx; // coeficiente beta 1 gorro
	beta0G = h_xb * (-beta1G) + h_yb ;	// coeficiente beta 0 gorro
	printf(" y = %f + %f x", beta0G, beta1G);
	return 0;
}