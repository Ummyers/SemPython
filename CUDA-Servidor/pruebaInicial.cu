/* Tarea 3 Seminario de Introducción a openMP-MPI y CUDA
*	Muñiz Patiño, Andrea Fernanda
*	Abril, Sem 2020-2 Semestre del coronavirus
*/
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include <sys/time.h>

/* Metodo ejecutado en el GPU que suma matrices
*	@m matriz a
*	@n matriz b
*	@p matriz c que contiene el resultado
*	@size el tamaño de la matriz
*/
__global__ void matrixSum(int* a,int* b, int* c, int size)
{
	// printf("ERROR en global\n");
	int pos = threadIdx.x;
	if(pos<size*size){
		c[pos] = a[pos] + b[pos];
	}
}

/* Metodo ejecutado en el GPU que multiplica matrices
*	@m matriz a
*	@n matriz b
*	@p matriz c que contiene el resultado
*	@size el tamaño de la matriz
*/
__global__ void matrixMult(int* m,int* n, int* p, int size)
{
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int p_sum;

	for(int i=0;i<size;i++){
		p_sum += m[row*size+i] * n[col*size+i];
	}
	p[row*size+col]=p_sum;
}

/* Metodo que inicializa las matrices con valores aleatorios
*	@a Matriz
*	@b Matriz
*	@n cantida de elementos de la matriz
*/

__host__ void inicializarMatriz(int* a, int* b, int n)
{
	srand(5);
	printf("Valor de tam: %d \n", n );
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			a[i*n+j]=rand() % 10 +1;
			b[i*n+j]=rand() % 10 +1;
		}
	}	
}

/*
* Método que imprime una matriz de tamaño @tam
* Un metodo tipo Host indica que solo puede ser llamado por
*  el mismo host (CPU)
*/
 __host__ void ImprimeMatriz(int tam, int* mat)
{
	for (int i = 0; i< tam; i++){
		for (int j = 0; j< tam; j++){
			printf("%d, ", mat[i*tam +j]);
		}
		printf(" \n");
	}
}

//Metodo que imprime solo la operación que se esta haciendo

__host__ void operacion(int a)
{
	switch(a){
		case 1:
			printf("*\n");
		break;
		case 2:
			printf("+\n");
		break;
	}
}

/* Metodo que hace copia de memoria del host al device y viceversa
*	@size tamaño del las matrices
*	@a puntero a primera matriz
* 	@b putero a segunda matriz
*	@res puntero a matriz solución
*	@op integer que indica que operación hará
*/

__host__ void addOnCuda(int size, const int* a, const int* b, int* res, int oper, int dim)
{
	//Declarando las variables del GPU
	int* dev_a = nullptr; //nullptr es null en puntero
	int* dev_b = nullptr;
	int* dev_res = nullptr;

	//Copiando en la memoria GPU las matrices. 
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_res, size);

    //Copiando los valores de las matrices al GPU desde host
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	
    //Dependiendo de la operación es la creación de Hilos y bloques

    //Para obtener el tiempo
    float tim = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Se hacen los calculos en GPU----------------
    hipEventRecord(start);
	
    //Opción de la multiplicación
    if(oper==1){
    	int threads_per_block = (int) dim/2;
    	int n = dim*dim; 
		dim3 block_size(threads_per_block,threads_per_block);
		dim3 grid_size(n / block_size.x, n / block_size.y);
		// llamar el kernel
		matrixMult<<< grid_size,block_size >>>(dev_a,dev_b,dev_res,dim);
   	//Opción de suma
   	}else{
   		int hilos = dim*dim;
		matrixSum<<<1,hilos>>>(dev_a,dev_b,dev_res,dim);
   	}

    //Se sincronizan los hilos para cuando terminen
    hipDeviceSynchronize();

   //Se termino de hacer los calculos
    hipEventRecord(stop);
    hipEventElapsedTime(&tim, start, stop);
    printf("El tiempo tomado por CUDA el algoritmo es de: %lf\n", tim);

    //Dirección destino, dir. origen, tamaño del dato, de donde a donde
    hipMemcpy(res, dev_res, size, hipMemcpyDeviceToHost);
}

int main(int argc, char const *argv[])
{
	int tamMatriz, opera;
	//matrices
	int* a;
	int* b;
	int* res;
	printf ("¿De qué tamaño es la matriz? \n");
	scanf ("%d", &tamMatriz);

	printf("¿Deseas multiplicar (1) o sumar (2) la matriz con valores aleatorios?\n");
	scanf("%d", &opera);

	//Validamos que el usuario ingrese una opción valida. 
	if((opera!=1)&&(opera!=2))
	{
		printf("Por favor indica un numero valido, 1 o 2.\n");
		return 0;
	}

	size_t bytes = tamMatriz*tamMatriz*sizeof(int);

	//Memoria para matrices
	a = (int*)malloc(bytes);
	b = (int*)malloc(bytes);
	res = (int*)malloc(bytes);

	//Llenamos la matriz
	inicializarMatriz(a, b, tamMatriz);

    //Para obtener el tiempo
    struct timeval inicio, fin;//nos permiten medir el tiempo de ejecucion
	gettimeofday(&inicio, NULL);//guarda el tiempo al inicio del programa
	int tiempo;
   
   //Se hacen los calculos en GPU----------------
    addOnCuda(bytes, a, b, res, opera, tamMatriz);
    
   //Se termino de hacer los calculos
    gettimeofday(&fin, NULL); //guarda el tiempo al final del programa
	tiempo = (fin.tv_sec - inicio.tv_sec)* 1000000 + (fin.tv_usec - inicio.tv_usec);
   

    //Se presentan los resultados de la operacion
    ImprimeMatriz(tamMatriz, a);
    operacion(opera);
    ImprimeMatriz(tamMatriz, b);
    printf("=\n");
    ImprimeMatriz(tamMatriz, res);
    //ImprimeMatriz(tamMatriz, res);

    printf("El tiempo tomado por el sistema del algoritmo es de:: %lf\n", tiempo);

    hipDeviceReset();
	return 0;
}