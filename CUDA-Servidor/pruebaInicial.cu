/* Tarea 3 Seminario de Introducción a openMP-MPI y CUDA
*	Muñiz Patiño, Andrea Fernanda
*	Abril, Sem 2020-2 Semestre del coronavirus
*/
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>


//multiplicacion de matrices basica 
__global__ void matrixMult(int* m,int* n, int* p, int size)
{
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int p_sum;

	for(int i=0;i<size;i++){
		p_sum += m[row*size+i] * n[col*size+i];
	}
	p[row*size+col]=p_sum;
}
/*
* Método que imprime una matriz de tamaño @tam
Un metodo tipo Host indica que solo puede ser llamado por
el mismo host (CPU)
*/
 __host__ void ImprimeMatriz(int tam, int* mat)
{
	printf("LOL\n");
// 	for (int i = 0; i < tam; i++)
// 	{
// 		for (int j = 0; j < tam; i++)
// 		{
// 			printf("LOL, ");
// 		}
// 		printf("\n");
// 	}
}

//Metodo que imprime solo la operación que se esta haciendo

__host__ void operacion(int a)
{
	switch(a){
		case 1:
			printf("+\n");
		break;
		case 2:
			printf("*\n");
		break;
	}
}

/* Metodo que hace copia de memoria del host al device 
*	@size tamaño del las matrices
*	@a puntero a primera matriz
* 	@b putero a segunda matriz
*	@res puntero a matriz solución
*/

__host__ void addOnCuda(int size, const int* a, const int* b, int* res)
{
	//Declarando las variables del GPU
	int* dev_a = nullptr; //nullptr es null en puntero
	int* dev_b = nullptr;
	int* dev_res = nullptr;

	//Copiando en la memoria GPU las matrices. 
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_res, size * sizeof(int));

    //Copiando los valores de las matrices al GPU desde host
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, size * sizeof(int), hipMemcpyHostToDevice);

    //Dirección destino, dir. origen, tamaño del dato, de donde a donde
    hipMemcpy(res, dev_res, size * sizeof(int), hipMemcpyDeviceToHost);
	
	//Libera memoria 
	hipFree(dev_res);
	hipFree(dev_a);
	hipFree(dev_b);
}

int main(int argc, char const *argv[])
{
	int tamMatriz;
	//matrices
	int* a;
	int* b;
	int* res;
	printf ("¿De qué tamaño es la matriz?");
	scanf ("%d", &tamMatriz);

	size_t bytes = tamMatriz*tamMatriz*sizeof(int);
	tamMatriz = tamMatriz * tamMatriz;

	//Memoria para matrices
	a = (int*)malloc(bytes);
	b = (int*)malloc(bytes);
	res = (int*)malloc(bytes);

    //Para obtener el tiempo
    float tim = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    //Se hace una copia de host a device
    addOnCuda(bytes, a, b, res);

    //Se hacen los calculos en GPU----------------
    hipEventRecord(start);


    //Se sincronizan los hilos para cuando terminen
    hipDeviceSynchronize();

   //Se termino de hacer los calculos
    hipEventRecord(stop);
    hipEventElapsedTime(&tim, start, stop);
    
    //Se presentan los resultados de la operacion
    ImprimeMatriz(tamMatriz, a);
    operacion(2);
    ImprimeMatriz(tamMatriz, b);
    printf("=\n");
    ImprimeMatriz(tamMatriz, res);

    printf("El tiempo tomado por el algoritmo es de: %lf\n", tim);
    hipDeviceReset();
	return 0;
}