#include "hip/hip_runtime.h"
/* Tarea 3 Seminario de Introducción a openMP-MPI y CUDA
*	Muñiz Patiño, Andrea Fernanda
*	Abril, Sem 2020-2 Semestre del coronavirus
*/
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>

//multiplicacion de matrices basica 
__global__ void matrixMult(int* m,int* n, int* p, int size)
{
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int p_sum;

	for(int i=0;i<size;i++){
		p_sum += m[row*size+i] * n[col*size+i];
	}
	p[row*size+col]=p_sum;
}
/*
* Método que imprime una matriz de tamaño @tam
Un metodo tipo Host indica que solo puede ser llamado por
el mismo host (CPU)
*/
__host__ void ImprimeMatriz(int tam, int* mat)
{
	for (int i = 0; i < tam; i++)
	{
		for (int j = 0; i < tam; i++)
		{
			printf("%d, ",mat[mat*tam+i][mat* tam +j]);
		}
		printf("\n");
	}
}

//Metodo que imprime solo la operación que se esta haciendo

__host__ void operacion(int a)
{
	switch(a){
		case 1:
			printf("+\n");
		break;
		case 2:
			printf("*\n");
		break;
	}
}

/* Metodo que hace copia de memoria del host al device 
*	@size tamaño del las matrices
*	@a puntero a primera matriz
* 	@b putero a segunda matriz
*	@res puntero a matriz solución
*/

__host__ void addOnCuda(int size, const int* a, const int* b, int* res)
{

	//Declarando las variables del GPU
	int* dev_a; //nullptr es null en puntero
	int* dev_b;
	int* dev_res;

	//Copiando en la memoria GPU las matrices. 
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_res, size * sizeof(int));

    //Copiando los valores de las matrices al GPU desde host
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, res, size * sizeof(int), hipMemcpyHostToDevice);
}

/* Metodo que extrae del GPU el resultado calculado
* @res puntero de donde se guardará el resultado de la operación
* @size tamaño del parametro res
*/
__host__ void extractFromCuda(int* res, int size){
	//Dirección destino, dir. origen, tamaño del dato, de donde a donde
    hipMemcpy(res, dev_res, size * sizeof(int), hipMemcpyDeviceToHost);
	
	//Libera memoria 
	hipFree(dev_res);
	hipFree(dev_a);
	hipFree(dev_b);
}

int main(int argc, char const *argv[])
{
	printf("Se ejecutara un ejemplo\n");
	const int arraySize = 3;
	//Matrices
    const int a[arraySize][arraySize] = {{1,2,3},{3,2,1},{3,1,2}};
    const int b[arraySize][arraySize] = {{4,5,6,},{6,5,4},{5,4,6}};
    //Resultado
    int res[arraySize][arraySize] = {{0}};

    //Se hace una copia de host a device
    addOnCuda(arraySize, a, b, res);

    //Se hacen los calculos en GPU----------------

    //Se sincronizan los hilos para cuando terminen
    hipDeviceSynchronize();

    //Se copia el resultado del device al host
    extractFromCuda(res, arraySize);

    //Se presentan los resultados de la operacion
    ImprimeMatriz(arraySize, a);
    operacion(2);
    ImprimeMatriz(arraySize, b);

    hipDeviceReset();
	return 0;
}